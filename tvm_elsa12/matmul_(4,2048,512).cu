
#include <hip/hip_runtime.h>
extern "C" __global__ void
default_function_kernel0(void *__restrict__ A, void *__restrict__ B,
                         void *__restrict__ compute) {
  float compute_local[2];
  __shared__ float A_shared[128];
  __shared__ float B_shared[4096];
  float A_shared_local[2];
  float B_shared_local[1];
  for (int i_c_init = 0; i_c_init < 2; ++i_c_init) {
    compute_local[(i_c_init)] = 0.000000e+00f;
  }
  for (int k_outer = 0; k_outer < 16; ++k_outer) {
    __syncthreads();
    for (int ax1_inner = 0; ax1_inner < 2; ++ax1_inner) {
      if (((int)threadIdx.x) < 32) {
        A_shared[((((((int)threadIdx.y) * 64) + (ax1_inner * 32)) +
                   ((int)threadIdx.x)))] =
            ((float *)A)[(((((((int)threadIdx.y) * 1024) + (ax1_inner * 512)) +
                            (k_outer * 32)) +
                           ((int)threadIdx.x)))];
      }
    }
    for (int ax1_inner1 = 0; ax1_inner1 < 64; ++ax1_inner1) {
      if (((int)threadIdx.x) < 32) {
        B_shared[((((((int)threadIdx.y) * 2048) + (ax1_inner1 * 32)) +
                   ((int)threadIdx.x)))] =
            ((float *)B)[((
                ((((((int)blockIdx.x) * 65536) + (((int)threadIdx.y) * 32768)) +
                  (ax1_inner1 * 512)) +
                 (k_outer * 32)) +
                ((int)threadIdx.x)))];
      }
    }
    __syncthreads();
    for (int k_inner = 0; k_inner < 32; ++k_inner) {
      for (int ax1 = 0; ax1 < 2; ++ax1) {
        A_shared_local[(ax1)] =
            A_shared[((((((int)threadIdx.y) * 64) + (ax1 * 32)) + k_inner))];
      }
      B_shared_local[(0)] = B_shared[(((((int)threadIdx.x) * 32) + k_inner))];
      for (int i_c = 0; i_c < 2; ++i_c) {
        compute_local[(i_c)] = (compute_local[(i_c)] +
                                (A_shared_local[(i_c)] * B_shared_local[(0)]));
      }
    }
  }
  for (int i_inner_inner = 0; i_inner_inner < 2; ++i_inner_inner) {
    ((float *)
         compute)[(((((((int)threadIdx.y) * 4096) + (i_inner_inner * 2048)) +
                     (((int)blockIdx.x) * 128)) +
                    ((int)threadIdx.x)))] = compute_local[(i_inner_inner)];
  }
}
