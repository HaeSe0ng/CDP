
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel0(void* __restrict__ A, void* __restrict__ B, void* __restrict__ T_dense) {
  float T_dense_rf[1];
  __shared__ float red_buf0[64];
  T_dense_rf[(0)] = 0.000000e+00f;
  for (int k_outer = 0; k_outer < 8; ++k_outer) {
    T_dense_rf[(0)] = (T_dense_rf[(0)] + (((float*)A)[((((((int)blockIdx.y) * 512) + (k_outer * 64)) + ((int)threadIdx.x)))] * ((float*)B)[((((((int)blockIdx.x) * 512) + (k_outer * 64)) + ((int)threadIdx.x)))]));
  }
  __syncthreads();
  ((volatile float*)red_buf0)[(((int)threadIdx.x))] = T_dense_rf[(0)];
  __syncthreads();
  if (((int)threadIdx.x) < 32) {
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 32))]);
  }
  __syncthreads();
  if (((int)threadIdx.x) < 16) {
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 16))]);
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 8))]);
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 4))]);
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 2))]);
    ((volatile float*)red_buf0)[(((int)threadIdx.x))] = (((volatile float*)red_buf0)[(((int)threadIdx.x))] + ((volatile float*)red_buf0)[((((int)threadIdx.x) + 1))]);
  }
  __syncthreads();
  if (((int)threadIdx.x) == 0) {
    ((float*)T_dense)[(((((int)blockIdx.y) * 16384) + ((int)blockIdx.x)))] = ((volatile float*)red_buf0)[(0)];
  }
}

