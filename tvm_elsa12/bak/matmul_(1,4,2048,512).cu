
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel0(void* __restrict__ A, void* __restrict__ B, void* __restrict__ compute) {
  float compute_local[4];
  __shared__ float A_shared[256];
  __shared__ float B_shared[256];
  float A_shared_local[4];
  float B_shared_local[1];
  for (int i_c_init = 0; i_c_init < 4; ++i_c_init) {
    compute_local[(i_c_init)] = 0.000000e+00f;
  }
  for (int k_outer = 0; k_outer < 8; ++k_outer) {
    __syncthreads();
    for (int ax1_inner = 0; ax1_inner < 4; ++ax1_inner) {
      for (int ax2_inner = 0; ax2_inner < 16; ++ax2_inner) {
        A_shared[((((ax1_inner * 64) + (((int)threadIdx.x) * 16)) + ax2_inner))] = ((float*)A)[(((((ax1_inner * 512) + (k_outer * 64)) + (((int)threadIdx.x) * 16)) + ax2_inner))];
      }
    }
    for (int ax1_inner1 = 0; ax1_inner1 < 4; ++ax1_inner1) {
      for (int ax2_inner1 = 0; ax2_inner1 < 16; ++ax2_inner1) {
        B_shared[((((ax1_inner1 * 64) + (((int)threadIdx.x) * 16)) + ax2_inner1))] = ((float*)B)[((((((((int)blockIdx.x) * 2048) + (ax1_inner1 * 512)) + (k_outer * 64)) + (((int)threadIdx.x) * 16)) + ax2_inner1))];
      }
    }
    __syncthreads();
    for (int k_inner = 0; k_inner < 64; ++k_inner) {
      #pragma unroll
      for (int ax1 = 0; ax1 < 4; ++ax1) {
        A_shared_local[(ax1)] = A_shared[(((ax1 * 64) + k_inner))];
      }
      B_shared_local[(0)] = B_shared[(((((int)threadIdx.x) * 64) + k_inner))];
      #pragma unroll
      for (int i_c = 0; i_c < 4; ++i_c) {
        compute_local[(i_c)] = (compute_local[(i_c)] + (A_shared_local[(i_c)] * B_shared_local[(0)]));
      }
    }
  }
  #pragma unroll
  for (int i_inner_inner = 0; i_inner_inner < 4; ++i_inner_inner) {
    ((float*)compute)[((((i_inner_inner * 2048) + (((int)blockIdx.x) * 4)) + ((int)threadIdx.x)))] = compute_local[(i_inner_inner)];
  }
}

