
#include <hip/hip_runtime.h>

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 700)
#define __shfl_sync(mask, var, lane, width) \
        __shfl((var), (lane), (width))

#define __shfl_down_sync(mask, var, offset, width) \
        __shfl_down((var), (offset), (width))

#define __shfl_up_sync(mask, var, offset, width) \
        __shfl_up((var), (offset), (width))
#endif

extern "C" __global__ void default_function_kernel0(void* __restrict__ A, void* __restrict__ B, void* __restrict__ T_dense) {
  float T_dense_rf[1];
  float red_buf0[1];
  T_dense_rf[(0)] = 0.000000e+00f;
  for (int k_outer = 0; k_outer < 16; ++k_outer) {
    T_dense_rf[(0)] = (T_dense_rf[(0)] + (((float*)A)[((((((int)blockIdx.y) * 512) + (k_outer * 32)) + ((int)threadIdx.x)))] * ((float*)B)[((((((int)blockIdx.x) * 512) + (k_outer * 32)) + ((int)threadIdx.x)))]));
  }
  unsigned int mask[1];
  float t0[1];
  red_buf0[(0)] = T_dense_rf[(0)];
  mask[(0)] = __activemask();
  t0[(0)] = __shfl_down_sync(mask[(0)], red_buf0[(0)], 16, 32);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __shfl_down_sync(mask[(0)], red_buf0[(0)], 8, 32);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __shfl_down_sync(mask[(0)], red_buf0[(0)], 4, 32);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __shfl_down_sync(mask[(0)], red_buf0[(0)], 2, 32);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  t0[(0)] = __shfl_down_sync(mask[(0)], red_buf0[(0)], 1, 32);
  red_buf0[(0)] = (red_buf0[(0)] + t0[(0)]);
  red_buf0[(0)] = __shfl_sync(mask[(0)], red_buf0[(0)], 0, 32);
  if (((int)threadIdx.x) == 0) {
    ((float*)T_dense)[(((((int)blockIdx.y) * 2048) + ((int)blockIdx.x)))] = red_buf0[(0)];
  }
}

