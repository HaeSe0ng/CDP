#include "hip/hip_runtime.h"
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>

#include <cstdlib>
#include <fstream>
#include <iostream>

#include "reduce_dev.cuh"
#include "seq2seq.h"
#include "torch_kernels.cuh"
#include "tvm_kernels.cuh"
#include "util.h"

__device__ int out_seq_len_d = 20;

extern __host__ __device__ void batch_matmul(float *A, float *B, float *C,
                                             int bsz, int M, int N, int K);

__device__ void argmax_dev(float *input_d, int64_t *output_d, int bsz,
                           int input_len) {
  int num_outputs = bsz;
  int inputs_per_output = input_len;

  static constexpr int vt0 = 4;
  auto ident =
      thrust::pair<float, int64_t>(at::numeric_limits<float>::lower_bound(), 0);

  auto config = ReduceConfig(16, num_outputs, inputs_per_output);
  config.set_block_dimension(inputs_per_output, num_outputs);
  int block_width = config.block_width;
  int block_height = config.block_height;
  config.input_mult[0] = config.split_input(block_width);
  if (config.values_per_thread() >= block_height * 16 ||
      config.values_per_thread() >= 256) {
    config.input_mult[1] = config.split_input(block_height);
  } else {
    config.output_mult[1] = config.split_output(block_height);
  }

  int num_reduce_dims = 1;
  int num_output_dims = 1;
  int64_t output_strides[2] = {0, sizeof(int64_t)};
  int64_t input_strides[2] = {sizeof(float),
                              (int64_t)sizeof(float) * inputs_per_output};
  int64_t *output_calc_strides[2] = {
      output_strides + num_reduce_dims,
      input_strides + num_reduce_dims,
  };
  int64_t *input_calc_strides[1] = {
      input_strides,
  };
  int64_t shape[2] = {inputs_per_output, num_outputs};
  auto output_calc = OffsetCalculator<2, uint32_t>(
      num_output_dims, shape + num_reduce_dims, &output_calc_strides[0]);
  auto input_calc = OffsetCalculator<1, uint32_t>(num_reduce_dims, shape,
                                                  &input_calc_strides[0]);

  auto reduce = ReduceOp<float, ArgMaxOps<float>, uint32_t, int64_t, vt0>(
      ArgMaxOps<float>{}, config, input_calc, output_calc,
      (const void *)input_d, (char *)output_d, nullptr, nullptr, nullptr, ident,
      1);
  reduce.accumulate = false;
  reduce.final_output = true;

  launch_reduce_kernel<ReduceConfig::MAX_NUM_THREADS>(config, reduce);
  /*
  hipError_t err = hipGetLastError();
  if (hipSuccess != err)
    printf("*cudaErr(%d) : %s \n", err, hipGetErrorString(err));
    */
}

__global__ void argmax_naive_kernel(float *input_d, int64_t *output_d, int bsz,
                                    int input_len) {
  float temp_topv, temp_v;
  int temp_topi;
  for (int b = 0; b < bsz; b++) {
    temp_topv = 0;
    temp_topi = 0;
    for (int vocab_idx = 0; vocab_idx < input_len; vocab_idx++) {
      temp_v = input_d[b * input_len + vocab_idx];
      if (temp_v > temp_topv) {
        temp_topi = vocab_idx;
        temp_topv = temp_v;
      }
    }
    output_d[b] = temp_topi;
    // printf("argmax] b=%d,temp_topi=%d,temp_topv=%f\n", b, temp_topi,
    // temp_topv);
  }
}
__host__ __device__ void lstm(float *input_d, float *hidden_d, float *w_ih_d,
                              float *w_hh_d, float *igate_d, float *hgate_d,
                              float *b_ih_d, float *b_hh_d, float *cell_d,
                              int bsz, int input_dim, int hidden_size,
                              int totalElements) {
  batch_matmul(input_d, w_ih_d, igate_d, 1, bsz, 4 * hidden_size,
               input_dim); // bsz, 4*hidden_size, input_dim
  batch_matmul(hidden_d, w_hh_d, hgate_d, 1, bsz, 4 * hidden_size,
               hidden_size); // bsz, 4*hidden_size, hidden_size
  lstm_cell_kernel<<<totalElements / AT_APPLY_THREADS_PER_BLOCK,
                     AT_APPLY_THREADS_PER_BLOCK>>>(
      igate_d, hgate_d, b_ih_d, b_hh_d, cell_d, hidden_d, cell_d, hidden_size,
      totalElements);
}
__host__ __device__ void embedding(int64_t *input_d, int emb_dim, int bsz,
                                   float *emb_tbl_d, float *emb_vec_d) {
  ptrdiff_t numIndices = bsz;

  ptrdiff_t outTotalSize = bsz * emb_dim;
  if (outTotalSize == 0) {
    return;
  }

  ptrdiff_t sliceSize = outTotalSize / numIndices;
  /*
  hipDeviceProp_t prop;
  int device;
  hipGetDevice(&device);
  hipGetDeviceProperties(&prop, device);
  */
  int mpc = 30; // prop.multiProcessorCount;

  // A reasonable choice for when to have each thread iterate over
  // indices to choose
  if (numIndices <= 16) {
    dim3 smallIndexGrid(
        MIN(THCCeilDiv(sliceSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
    dim3 smallIndexBlock(MIN(sliceSize, (ptrdiff_t)128));
    indexSelectSmallIndex<<<smallIndexGrid, smallIndexBlock, 0>>>(
        emb_vec_d, emb_tbl_d, input_d, sliceSize, numIndices);
  } else {
    dim3 largeIndexGrid(
        MIN(THCCeilDiv(outTotalSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
    dim3 largeIndexBlock(MIN(outTotalSize, (ptrdiff_t)128));
    indexSelectLargeIndex<<<largeIndexGrid, largeIndexBlock, 0>>>(
        emb_vec_d, emb_tbl_d, input_d, outTotalSize, sliceSize);
  }
}

__device__ void argmax_naive_dev(float *input, int64_t *output, int bsz,
                                 int input_len) {
  argmax_naive_kernel<<<1, 1>>>(input, output, bsz, input_len);
}
void seq2seq_encode(int64_t *input_d, float *emb_tbl_d, float *emb_vec_d,
                    float *hidden_d, float *w_ih_d, float *w_hh_d,
                    float *igate_d, float *hgate_d, float *b_ih_d,
                    float *b_hh_d, float *cell_d, float *w_ho_d, int bsz,
                    int emb_dim, int hidden_size, int totalElements,
                    int seq_length) {
  for (int i = 0; i < seq_length; i++) {
    embedding(input_d + i * bsz, emb_dim, bsz, emb_tbl_d, emb_vec_d);

    lstm(emb_vec_d, hidden_d, w_ih_d, w_hh_d, igate_d, hgate_d, b_ih_d, b_hh_d,
         cell_d, bsz, emb_dim, hidden_size, totalElements);
  }
}

__global__ void seq2seq_decode(
    float *emb_tbl_d, float *emb_vec_d, float *hidden_d, float *w_ih_d,
    float *w_hh_d, float *igate_d, float *hgate_d, float *b_ih_d, float *b_hh_d,
    float *cell_d, float *output_onehot_d, float *w_ho_d, int64_t *output_d,
    int64_t *output, int64_t *eos_d, int bsz, int emb_dim, int hidden_size,
    int totalElements, int tgt_vocab_size, int max_len, int64_t *sos_batch_d) {
  int i;
  bool is_end;

  for (i = 0; i < 16; i++) {
    is_end = true;
    if (i == 0)
      embedding(sos_batch_d, emb_dim, bsz, emb_tbl_d, emb_vec_d);
    else
      embedding(output_d + bsz * (i - 1), emb_dim, bsz, emb_tbl_d, emb_vec_d);
    lstm(emb_vec_d, hidden_d, w_ih_d, w_hh_d, igate_d, hgate_d, b_ih_d, b_hh_d,
         cell_d, bsz, emb_dim, hidden_size, totalElements);
    batch_matmul(hidden_d, w_ho_d, output_onehot_d + bsz * tgt_vocab_size * i,
                 1, bsz, tgt_vocab_size,
                 hidden_size); // bsz, tgt_vocab_size, hidden_size
    argmax_dev(output_onehot_d + bsz * tgt_vocab_size * i, output_d + bsz * i,
               bsz, tgt_vocab_size);
    hipDeviceSynchronize();
    //__syncthreads();
    for (int b = 0; b < bsz; b++) {
      // printf("i=%d, output_d[%d]=%d, eos_d[%d]=%d\n", i, bsz * i + b,
      //      output_d[bsz * i + b], b, eos_d[b]);
      if (output_d[bsz * i + b] != eos_d[b]) {
        is_end = false;
        break;
      }
    }
    if (is_end) {
      i++;
      break;
    }
  }
  // printf("end: out_seq_len=%d\n", i);
  out_seq_len_d = i;
}
int seq2seq_inf(int64_t *input, int64_t *output, int64_t sos, int64_t *eos,
                int emb_dim, int seq_length, int hidden_size, int batch_size,
                int src_vocab_size, int tgt_vocab_size, int max_len,
                float *res) {
  int64_t *input_d;
  hipMalloc((void **)&input_d, sizeof(int64_t) * seq_length * batch_size);
  hipMemcpy(input_d, input, (sizeof(int64_t) * seq_length * batch_size),
             hipMemcpyHostToDevice);

  hipHostMalloc((void **)&output, sizeof(int64_t) * batch_size * max_len);

  int64_t *sos_batch_d;
  hipMalloc((void **)&sos_batch_d, sizeof(int64_t) * batch_size);
  hipMemset(sos_batch_d, sos, sizeof(int64_t) * batch_size);

  int totalElements = batch_size * hidden_size;
  int64_t *output_d, *eos_d;
  float *w_ih_enc, *w_hh_enc, *b_ih_enc, *b_hh_enc;
  float *w_ih_dec, *w_hh_dec, *b_ih_dec, *b_hh_dec, *w_ho;
  float *emb_tbl_enc, *emb_tbl_dec;
  float *hidden_d, *igate_d, *hgate_d, *cell_d;
  float *w_ih_enc_d, *w_hh_enc_d, *b_ih_enc_d, *b_hh_enc_d;
  float *w_ih_dec_d, *w_hh_dec_d, *b_ih_dec_d, *b_hh_dec_d, *w_ho_d;
  float *output_onehot_d, *emb_tbl_enc_d, *emb_tbl_dec_d, *emb_vec_d;

  hipMalloc((void **)&eos_d, sizeof(int64_t) * batch_size);
  hipMemcpy(eos_d, eos, (sizeof(int64_t) * batch_size),
             hipMemcpyHostToDevice);

  alloc_rand_mat<float>(&emb_tbl_enc, src_vocab_size, emb_dim);
  hipMalloc((void **)&emb_tbl_enc_d, sizeof(float) * src_vocab_size * emb_dim);
  alloc_rand_mat<float>(&emb_tbl_dec, tgt_vocab_size, emb_dim);
  hipMalloc((void **)&emb_tbl_dec_d, sizeof(float) * tgt_vocab_size * emb_dim);
  hipMalloc((void **)&emb_vec_d, sizeof(float) * batch_size * emb_dim);

  hipMemcpy(emb_tbl_enc_d, emb_tbl_enc,
             (sizeof(float) * src_vocab_size * emb_dim),
             hipMemcpyHostToDevice);
  hipMemcpy(emb_tbl_dec_d, emb_tbl_dec,
             (sizeof(float) * tgt_vocab_size * emb_dim),
             hipMemcpyHostToDevice);

  hipMalloc((void **)&output_onehot_d,
             sizeof(float) * batch_size * tgt_vocab_size * max_len);
  hipMalloc((void **)&output_d, sizeof(int64_t) * batch_size * max_len);

  hipMalloc((void **)&hidden_d, sizeof(float) * batch_size * hidden_size);
  hipMalloc((void **)&cell_d, sizeof(float) * batch_size * hidden_size);

  hipMalloc((void **)&igate_d, sizeof(float) * batch_size * (4 * hidden_size));
  hipMalloc((void **)&hgate_d, sizeof(float) * batch_size * (4 * hidden_size));

  hipMalloc((void **)&w_ih_enc_d, sizeof(float) * (4 * hidden_size) * emb_dim);
  hipMalloc((void **)&w_hh_enc_d,
             sizeof(float) * (4 * hidden_size) * hidden_size);
  hipMalloc((void **)&b_ih_enc_d, sizeof(float) * 4 * hidden_size);
  hipMalloc((void **)&b_hh_enc_d, sizeof(float) * 4 * hidden_size);

  hipMalloc((void **)&w_ih_dec_d, sizeof(float) * (4 * hidden_size) * emb_dim);
  hipMalloc((void **)&w_hh_dec_d,
             sizeof(float) * (4 * hidden_size) * hidden_size);
  hipMalloc((void **)&b_ih_dec_d, sizeof(float) * 4 * hidden_size);
  hipMalloc((void **)&b_hh_dec_d, sizeof(float) * 4 * hidden_size);
  hipMalloc((void **)&w_ho_d, sizeof(float) * tgt_vocab_size * hidden_size);

  alloc_rand_mat<float>(&w_ih_enc, (4 * hidden_size), emb_dim);
  alloc_rand_mat<float>(&w_hh_enc, (4 * hidden_size), hidden_size);
  alloc_rand_mat<float>(&b_ih_enc, 1, 4 * hidden_size);
  alloc_rand_mat<float>(&b_hh_enc, 1, 4 * hidden_size);

  hipMemcpy(w_ih_enc_d, w_ih_enc,
             (sizeof(float) * emb_dim * (4 * hidden_size)),
             hipMemcpyHostToDevice);
  hipMemcpy(w_hh_enc_d, w_hh_enc,
             (sizeof(float) * hidden_size * (4 * hidden_size)),
             hipMemcpyHostToDevice);
  hipMemcpy(b_ih_enc_d, b_ih_enc, (sizeof(float) * 4 * hidden_size),
             hipMemcpyHostToDevice);
  hipMemcpy(b_hh_enc_d, b_hh_enc, (sizeof(float) * 4 * hidden_size),
             hipMemcpyHostToDevice);

  alloc_rand_mat<float>(&w_ih_dec, (4 * hidden_size), emb_dim);
  alloc_rand_mat<float>(&w_hh_dec, (4 * hidden_size), hidden_size);
  alloc_rand_mat<float>(&b_ih_dec, 1, 4 * hidden_size);
  alloc_rand_mat<float>(&b_hh_dec, 1, 4 * hidden_size);
  hipMemcpy(w_ih_dec_d, w_ih_dec,
             (sizeof(float) * emb_dim * (4 * hidden_size)),
             hipMemcpyHostToDevice);
  hipMemcpy(w_hh_dec_d, w_hh_dec,
             (sizeof(float) * hidden_size * (4 * hidden_size)),
             hipMemcpyHostToDevice);
  hipMemcpy(b_ih_dec_d, b_ih_dec, (sizeof(float) * 4 * hidden_size),
             hipMemcpyHostToDevice);
  hipMemcpy(b_hh_dec_d, b_hh_dec, (sizeof(float) * 4 * hidden_size),
             hipMemcpyHostToDevice);

  alloc_rand_mat<float>(&w_ho, tgt_vocab_size, hidden_size);
  hipMemcpy(w_ho_d, w_ho, (sizeof(float) * hidden_size * tgt_vocab_size),
             hipMemcpyHostToDevice);

  float elapsed_time, elapsed_time_enc = 0, elapsed_time_dec = 0,
                      elapsed_time_mem = 0;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  int out_seq_len;
  bool prof = true;

  if (prof) {
    int num_itr = 200;
    for (int i = -num_itr; i < num_itr; i++) {
      hipMemset(hidden_d, 0, sizeof(float) * batch_size * hidden_size);
      hipMemset(cell_d, 0, sizeof(float) * batch_size * hidden_size);

      if (i < 0) {
        seq2seq_encode(input_d, emb_tbl_enc_d, emb_vec_d, hidden_d, w_ih_enc_d,
                       w_hh_enc_d, igate_d, hgate_d, b_ih_enc_d, b_hh_enc_d,
                       cell_d, w_ho_d, batch_size, emb_dim, hidden_size,
                       totalElements, seq_length);
        seq2seq_decode<<<1, 1>>>(
            emb_tbl_dec_d, emb_vec_d, hidden_d, w_ih_dec_d, w_hh_dec_d, igate_d,
            hgate_d, b_ih_dec_d, b_hh_dec_d, cell_d, output_onehot_d, w_ho_d,
            output_d, output, eos_d, batch_size, emb_dim, hidden_size,
            totalElements, tgt_vocab_size, max_len, sos_batch_d);
      } else {

        hipEventRecord(start);

        seq2seq_encode(input_d, emb_tbl_enc_d, emb_vec_d, hidden_d, w_ih_enc_d,
                       w_hh_enc_d, igate_d, hgate_d, b_ih_enc_d, b_hh_enc_d,
                       cell_d, w_ho_d, batch_size, emb_dim, hidden_size,
                       totalElements, seq_length);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed_time, start, stop);
        elapsed_time_enc += elapsed_time;

        hipEventRecord(start);

        seq2seq_decode<<<1, 1>>>(
            emb_tbl_dec_d, emb_vec_d, hidden_d, w_ih_dec_d, w_hh_dec_d, igate_d,
            hgate_d, b_ih_dec_d, b_hh_dec_d, cell_d, output_onehot_d, w_ho_d,
            output_d, output, eos_d, batch_size, emb_dim, hidden_size,
            totalElements, tgt_vocab_size, max_len, sos_batch_d);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed_time, start, stop);
        elapsed_time_dec += elapsed_time;

        hipEventRecord(start);
        hipMemcpyFromSymbolAsync(&out_seq_len, HIP_SYMBOL(out_seq_len_d),
                                  sizeof(out_seq_len), 0,
                                  hipMemcpyDeviceToHost);
        hipMemcpyAsync(output, output_d,
                        (sizeof(int64_t) * batch_size * out_seq_len),
                        hipMemcpyDeviceToHost);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed_time, start, stop);
        elapsed_time_mem += elapsed_time;
      }
    }
    elapsed_time_enc /= num_itr;
    elapsed_time_dec /= num_itr;
    elapsed_time_mem /= num_itr;
    res[0] = elapsed_time_enc;
    res[1] = elapsed_time_dec + elapsed_time_mem;
    res[2] = elapsed_time_mem;

  }
  // one time execution
  else {
    hipMemset(hidden_d, 0, sizeof(float) * batch_size * hidden_size);
    hipMemset(cell_d, 0, sizeof(float) * batch_size * hidden_size);
    seq2seq_encode(input_d, emb_tbl_enc_d, emb_vec_d, hidden_d, w_ih_enc_d,
                   w_hh_enc_d, igate_d, hgate_d, b_ih_enc_d, b_hh_enc_d, cell_d,
                   w_ho_d, batch_size, emb_dim, hidden_size, totalElements,
                   seq_length);
    seq2seq_decode<<<1, 1>>>(
        emb_tbl_dec_d, emb_vec_d, hidden_d, w_ih_dec_d, w_hh_dec_d, igate_d,
        hgate_d, b_ih_dec_d, b_hh_dec_d, cell_d, output_onehot_d, w_ho_d,
        output_d, output, eos_d, batch_size, emb_dim, hidden_size,
        totalElements, tgt_vocab_size, max_len, sos_batch_d);
    hipMemcpyFromSymbol(&out_seq_len, HIP_SYMBOL(out_seq_len_d), sizeof(out_seq_len), 0,
                         hipMemcpyDeviceToHost);
    // out_seq_len = 16;
    hipMemcpy(output, output_d, (sizeof(int64_t) * batch_size * out_seq_len),
               hipMemcpyDeviceToHost);
  }

  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipFree(input_d);
  hipFree(sos_batch_d);
  hipHostFree(output);
  hipFree(emb_tbl_enc_d);
  hipFree(emb_tbl_dec_d);
  hipFree(output_onehot_d);
  hipFree(output_d);
  hipFree(hidden_d);
  hipFree(cell_d);
  hipFree(igate_d);
  hipFree(hgate_d);
  hipFree(w_ih_enc_d);
  hipFree(w_hh_enc_d);
  hipFree(b_ih_enc_d);
  hipFree(b_hh_enc_d);
  hipFree(w_ih_dec_d);
  hipFree(w_hh_dec_d);
  hipFree(b_ih_dec_d);
  hipFree(b_hh_dec_d);
  hipFree(w_ho_d);

  free(w_ih_enc);
  free(w_hh_enc);
  free(b_ih_enc);
  free(b_hh_enc);
  free(w_ih_dec);
  free(w_hh_dec);
  free(b_ih_dec);
  free(b_hh_dec);
  free(w_ho);
  free(emb_tbl_enc);
  free(emb_tbl_dec);

  return 0;
}

__host__ __device__ void batch_matmul(float *A, float *B, float *C, int bsz,
                                      int M, int N, int K) {
  auto cfg = matmul_kernel_launch_cfg(bsz, M, N, K);
  (*(cfg.func))<<<cfg.gridDim, cfg.blockDim>>>(A, B, C);
}