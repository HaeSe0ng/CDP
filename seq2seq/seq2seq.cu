#include "hip/hip_runtime.h"
#include "tvm_kernels.cuh"
#include <cuda_device_runtime_api.h>

#include <stdio.h>

#include <cstdlib>
#include <fstream>
#include <iostream>

#include "seq2seq.h"
#include "util.h"

#define AT_APPLY_THREADS_PER_BLOCK 512

__device__ int out_seq_len_d = 20;

void batch_matmul(float *A, float *B, float *C, int bsz, int M, int N, int K);
__device__ void batch_matmul_dev(float *A, float *B, float *C, int bsz, int M,
                                 int N, int K);

template <typename T> __device__ __forceinline__ T sigmoid(T in) {
  T one = static_cast<T>(1.0);
  return one / (one + exp(-in));
}

__global__ void matmul_kernel(float *A, float *B, float *C, int M, int N,
                              int K) {}
// bias1: input_bias, bias2: hidden_bias, cx: last cell state, hsz: hidden_size
__global__ void lstm_cell_kernel(float *input, float *hidden, float *bias1,
                                 float *bias2, float *_cx, float *_hy,
                                 float *_cy, int hsz, int totalElements) {
  for (int linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
       linearIndex < totalElements; linearIndex += gridDim.x * blockDim.x) {
    int offset = (linearIndex / hsz) * 4 * hsz + linearIndex % hsz;

    float iig = input[offset + 0 * hsz];
    float ifg = input[offset + 1 * hsz];
    float icg = input[offset + 2 * hsz];
    float iog = input[offset + 3 * hsz];

    float hig = hidden[offset + 0 * hsz];
    float hfg = hidden[offset + 1 * hsz];
    float hcg = hidden[offset + 2 * hsz];
    float hog = hidden[offset + 3 * hsz];

    float cx = _cx[linearIndex];

    float *hy = &_hy[linearIndex];
    float *cy = &_cy[linearIndex];

    float b1i, b1f, b1c, b1o;
    float b2i, b2f, b2c, b2o;

    b1i = bias1[linearIndex % hsz + 0 * hsz];
    b1f = bias1[linearIndex % hsz + 1 * hsz];
    b1c = bias1[linearIndex % hsz + 2 * hsz];
    b1o = bias1[linearIndex % hsz + 3 * hsz];

    b2i = bias2[linearIndex % hsz + 0 * hsz];
    b2f = bias2[linearIndex % hsz + 1 * hsz];
    b2c = bias2[linearIndex % hsz + 2 * hsz];
    b2o = bias2[linearIndex % hsz + 3 * hsz];

    float ig, fg, cg, og;
    float f_hy, f_cy;

    ig = sigmoid(iig + hig + b1i + b2i);
    fg = sigmoid(ifg + hfg + b1f + b2f);
    cg = tanh(icg + hcg + b1c + b2c);
    og = sigmoid(iog + hog + b1o + b2o);

    f_cy = (fg * cx) + (ig * cg);
    f_hy = og * tanh(f_cy);

    *hy = f_hy;
    *cy = f_cy;
  }
}
__global__ void argmax_kernel(float *input_d, int *output_d, int bsz,
                              int input_len) {
  float temp_topv, temp_v;
  int temp_topi;
  for (int b = 0; b < bsz; b++) {
    temp_topv = 0;
    temp_topi = 0;
    for (int vocab_idx = 0; vocab_idx < input_len; vocab_idx++) {
      temp_v = input_d[b * input_len + vocab_idx];
      if (temp_v > temp_topv) {
        temp_topi = vocab_idx;
        temp_topv = temp_v;
      }
    }
    output_d[b] = temp_topi;
    // printf("argmax] b=%d,temp_topi=%d,temp_topv=%f\n", b, temp_topi,
    // temp_topv);
  }
}
void lstm(float *input_d, float *hidden_d, float *w_ih_d, float *w_hh_d,
          float *igate_d, float *hgate_d, float *b_ih_d, float *b_hh_d,
          float *cell_d, int bsz, int input_dim, int hidden_size,
          int totalElements) {
  batch_matmul(input_d, w_ih_d, igate_d, 1, bsz, 4 * hidden_size,
               input_dim); // bsz, 4*hidden_size, input_dim
  batch_matmul(hidden_d, w_hh_d, hgate_d, 1, bsz, 4 * hidden_size,
               hidden_size); // bsz, 4*hidden_size, hidden_size
  lstm_cell_kernel<<<totalElements / AT_APPLY_THREADS_PER_BLOCK,
                     AT_APPLY_THREADS_PER_BLOCK>>>(
      igate_d, hgate_d, b_ih_d, b_hh_d, cell_d, hidden_d, cell_d, hidden_size,
      totalElements);
}
__device__ void lstm_dev(float *input_d, float *hidden_d, float *w_ih_d,
                         float *w_hh_d, float *igate_d, float *hgate_d,
                         float *b_ih_d, float *b_hh_d, float *cell_d, int bsz,
                         int input_dim, int hidden_size, int totalElements) {
  batch_matmul_dev(input_d, w_ih_d, igate_d, 1, bsz, 4 * hidden_size,
                   input_dim); // bsz, 4*hidden_size, input_dim
  batch_matmul_dev(hidden_d, w_hh_d, hgate_d, 1, bsz, 4 * hidden_size,
                   hidden_size); // bsz, 4*hidden_size, hidden_size
  lstm_cell_kernel<<<totalElements / AT_APPLY_THREADS_PER_BLOCK,
                     AT_APPLY_THREADS_PER_BLOCK>>>(
      igate_d, hgate_d, b_ih_d, b_hh_d, cell_d, hidden_d, cell_d, hidden_size,
      totalElements);
}
void embedding(int *input, int emb_dim, int bsz, float *emb_tbl_d,
               float *emb_vec_d) {
  for (int b = 0; b < bsz; b++) {
    hipMemcpy(emb_vec_d + b * emb_dim, emb_tbl_d + input[b] * emb_dim,
               (sizeof(float) * emb_dim), hipMemcpyDeviceToDevice);
  }
}
__device__ void embedding_dev(int *input_d, int emb_dim, int bsz,
                              float *emb_tbl_d, float *emb_vec_d) {
  for (int b = 0; b < bsz; b++) {
    memcpy(emb_vec_d + b * emb_dim, emb_tbl_d + input_d[b] * emb_dim,
           (sizeof(float) * emb_dim));
  }
}
__device__ void argmax_dev(float *input, int *output, int bsz, int input_len) {
  argmax_kernel<<<1, 1>>>(input, output, bsz, input_len);
}
void seq2seq_encode(int *input, float *emb_tbl_d, float *emb_vec_d,
                    float *hidden_d, float *w_ih_d, float *w_hh_d,
                    float *igate_d, float *hgate_d, float *b_ih_d,
                    float *b_hh_d, float *cell_d, float *w_ho_d, int bsz,
                    int emb_dim, int hidden_size, int totalElements,
                    int seq_length) {
  for (int i = 0; i < seq_length; i++) {
    embedding(input + i * bsz, emb_dim, bsz, emb_tbl_d, emb_vec_d);

    lstm(emb_vec_d, hidden_d, w_ih_d, w_hh_d, igate_d, hgate_d, b_ih_d, b_hh_d,
         cell_d, bsz, emb_dim, hidden_size, totalElements);
  }
}

__global__ void seq2seq_decode(
    float *emb_tbl_d, float *emb_vec_d, float *hidden_d, float *w_ih_d,
    float *w_hh_d, float *igate_d, float *hgate_d, float *b_ih_d, float *b_hh_d,
    float *cell_d, float *output_onehot_d, float *w_ho_d, int *output_d,
    int *output, int *eos_d, int bsz, int emb_dim, int hidden_size,
    int totalElements, int tgt_vocab_size, int max_len, int *sos_batch_d) {
  int i;
  bool is_end;
  for (i = 0; i < max_len; i++) {
    is_end = true;
    if (i == 0)
      embedding_dev(sos_batch_d, emb_dim, bsz, emb_tbl_d, emb_vec_d);
    else
      embedding_dev(output_d + bsz * (i - 1), emb_dim, bsz, emb_tbl_d,
                    emb_vec_d);
    lstm_dev(emb_vec_d, hidden_d, w_ih_d, w_hh_d, igate_d, hgate_d, b_ih_d,
             b_hh_d, cell_d, bsz, emb_dim, hidden_size, totalElements);
    batch_matmul_dev(hidden_d, w_ho_d,
                     output_onehot_d + bsz * tgt_vocab_size * i, 1, bsz,
                     tgt_vocab_size,
                     hidden_size); // bsz, tgt_vocab_size, hidden_size
    argmax_dev(output_onehot_d + bsz * tgt_vocab_size * i, output_d + bsz * i,
               bsz, tgt_vocab_size);
    hipDeviceSynchronize();
    //__syncthreads();
    for (int b = 0; b < bsz; b++) {
      // printf("i=%d, output_d[%d]=%d, eos_d[%d]=%d\n", i, bsz * i + b,
      //       output_d[bsz * i + b], b, eos_d[b]);
      if (output_d[bsz * i + b] != eos_d[b]) {
        is_end = false;
        break;
      }
    }
    if (is_end) {
      printf("end: out_seq_len=%d\n", i + 1);
      out_seq_len_d = i + 1;
      break;
    }
  }
}
int seq2seq_inf(int *input, int *output, int sos, int *eos, int emb_dim,
                int seq_length, int hidden_size, int batch_size,
                int src_vocab_size, int tgt_vocab_size, int max_len) {
  hipHostMalloc((void **)&output, sizeof(int) * batch_size * max_len);
  int *sos_batch_d;
  hipMalloc((void **)&sos_batch_d, sizeof(int) * batch_size);
  hipMemset(sos_batch_d, sos, sizeof(int) * batch_size);

  int totalElements = batch_size * hidden_size;
  int *output_d, *eos_d;
  float *w_ih_enc, *w_hh_enc, *b_ih_enc, *b_hh_enc;
  float *w_ih_dec, *w_hh_dec, *b_ih_dec, *b_hh_dec, *w_ho;
  float *emb_tbl_enc, *emb_tbl_dec;
  float *hidden_d, *igate_d, *hgate_d, *cell_d;
  float *w_ih_enc_d, *w_hh_enc_d, *b_ih_enc_d, *b_hh_enc_d;
  float *w_ih_dec_d, *w_hh_dec_d, *b_ih_dec_d, *b_hh_dec_d, *w_ho_d;
  float *output_onehot_d, *emb_tbl_enc_d, *emb_tbl_dec_d, *emb_vec_d;

  hipMalloc((void **)&eos_d, sizeof(int) * batch_size);
  hipMemcpy(eos_d, eos, (sizeof(int) * batch_size), hipMemcpyHostToDevice);

  alloc_rand_mat<float>(&emb_tbl_enc, src_vocab_size, emb_dim);
  hipMalloc((void **)&emb_tbl_enc_d, sizeof(float) * src_vocab_size * emb_dim);
  alloc_rand_mat<float>(&emb_tbl_dec, tgt_vocab_size, emb_dim);
  hipMalloc((void **)&emb_tbl_dec_d, sizeof(float) * tgt_vocab_size * emb_dim);
  hipMalloc((void **)&emb_vec_d, sizeof(float) * batch_size * emb_dim);

  hipMemcpy(emb_tbl_enc_d, emb_tbl_enc,
             (sizeof(float) * src_vocab_size * emb_dim),
             hipMemcpyHostToDevice);
  hipMemcpy(emb_tbl_dec_d, emb_tbl_dec,
             (sizeof(float) * tgt_vocab_size * emb_dim),
             hipMemcpyHostToDevice);

  hipMalloc((void **)&output_onehot_d,
             sizeof(float) * batch_size * tgt_vocab_size * max_len);
  hipMalloc((void **)&output_d, sizeof(int) * batch_size * max_len);

  hipMalloc((void **)&hidden_d, sizeof(float) * batch_size * hidden_size);
  hipMalloc((void **)&cell_d, sizeof(float) * batch_size * hidden_size);

  hipMalloc((void **)&igate_d, sizeof(float) * batch_size * (4 * hidden_size));
  hipMalloc((void **)&hgate_d, sizeof(float) * batch_size * (4 * hidden_size));

  hipMalloc((void **)&w_ih_enc_d, sizeof(float) * (4 * hidden_size) * emb_dim);
  hipMalloc((void **)&w_hh_enc_d,
             sizeof(float) * (4 * hidden_size) * hidden_size);
  hipMalloc((void **)&b_ih_enc_d, sizeof(float) * 4 * hidden_size);
  hipMalloc((void **)&b_hh_enc_d, sizeof(float) * 4 * hidden_size);

  hipMalloc((void **)&w_ih_dec_d, sizeof(float) * (4 * hidden_size) * emb_dim);
  hipMalloc((void **)&w_hh_dec_d,
             sizeof(float) * (4 * hidden_size) * hidden_size);
  hipMalloc((void **)&b_ih_dec_d, sizeof(float) * 4 * hidden_size);
  hipMalloc((void **)&b_hh_dec_d, sizeof(float) * 4 * hidden_size);
  hipMalloc((void **)&w_ho_d, sizeof(float) * tgt_vocab_size * hidden_size);

  hipMemset(hidden_d, 0, sizeof(float) * batch_size * hidden_size);
  hipMemset(cell_d, 0, sizeof(float) * batch_size * hidden_size);
  alloc_rand_mat<float>(&w_ih_enc, (4 * hidden_size), emb_dim);
  alloc_rand_mat<float>(&w_hh_enc, (4 * hidden_size), hidden_size);
  alloc_rand_mat<float>(&b_ih_enc, 1, 4 * hidden_size);
  alloc_rand_mat<float>(&b_hh_enc, 1, 4 * hidden_size);

  hipMemcpy(w_ih_enc_d, w_ih_enc,
             (sizeof(float) * emb_dim * (4 * hidden_size)),
             hipMemcpyHostToDevice);
  hipMemcpy(w_hh_enc_d, w_hh_enc,
             (sizeof(float) * hidden_size * (4 * hidden_size)),
             hipMemcpyHostToDevice);
  hipMemcpy(b_ih_enc_d, b_ih_enc, (sizeof(float) * 4 * hidden_size),
             hipMemcpyHostToDevice);
  hipMemcpy(b_hh_enc_d, b_hh_enc, (sizeof(float) * 4 * hidden_size),
             hipMemcpyHostToDevice);

  alloc_rand_mat<float>(&w_ih_dec, (4 * hidden_size), emb_dim);
  alloc_rand_mat<float>(&w_hh_dec, (4 * hidden_size), hidden_size);
  alloc_rand_mat<float>(&b_ih_dec, 1, 4 * hidden_size);
  alloc_rand_mat<float>(&b_hh_dec, 1, 4 * hidden_size);
  hipMemcpy(w_ih_dec_d, w_ih_dec,
             (sizeof(float) * emb_dim * (4 * hidden_size)),
             hipMemcpyHostToDevice);
  hipMemcpy(w_hh_dec_d, w_hh_dec,
             (sizeof(float) * hidden_size * (4 * hidden_size)),
             hipMemcpyHostToDevice);
  hipMemcpy(b_ih_dec_d, b_ih_dec, (sizeof(float) * 4 * hidden_size),
             hipMemcpyHostToDevice);
  hipMemcpy(b_hh_dec_d, b_hh_dec, (sizeof(float) * 4 * hidden_size),
             hipMemcpyHostToDevice);

  alloc_rand_mat<float>(&w_ho, tgt_vocab_size, hidden_size);
  hipMemcpy(w_ho_d, w_ho, (sizeof(float) * hidden_size * tgt_vocab_size),
             hipMemcpyHostToDevice);

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  // main logic

  seq2seq_encode(input, emb_tbl_enc_d, emb_vec_d, hidden_d, w_ih_enc_d,
                 w_hh_enc_d, igate_d, hgate_d, b_ih_enc_d, b_hh_enc_d, cell_d,
                 w_ho_d, batch_size, emb_dim, hidden_size, totalElements,
                 seq_length);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float elapsed_time;
  hipEventElapsedTime(&elapsed_time, start, stop);
  printf("[CDP_encode]execution time: %fms\n", elapsed_time);
  hipEventRecord(start);
  int out_seq_len;
  seq2seq_decode<<<1, 1>>>(emb_tbl_dec_d, emb_vec_d, hidden_d, w_ih_dec_d,
                           w_hh_dec_d, igate_d, hgate_d, b_ih_dec_d, b_hh_dec_d,
                           cell_d, output_onehot_d, w_ho_d, output_d, output,
                           eos_d, batch_size, emb_dim, hidden_size,
                           totalElements, tgt_vocab_size, max_len, sos_batch_d);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time, start, stop);
  printf("[CDP_decode]execution time: %fms\n", elapsed_time);

  hipEventRecord(start);

  hipMemcpyFromSymbol(&out_seq_len, HIP_SYMBOL(out_seq_len_d), sizeof(out_seq_len), 0,
                       hipMemcpyDeviceToHost);
  hipMemcpy(output, output_d, (sizeof(int) * batch_size * out_seq_len),
             hipMemcpyDeviceToHost);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time, start, stop);
  printf("[CDP_decode_memcpy]execution time: %fms\n", elapsed_time);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipFree(emb_tbl_enc_d);
  hipFree(emb_tbl_dec_d);
  hipFree(output_onehot_d);
  hipFree(output_d);
  hipFree(hidden_d);
  hipFree(cell_d);
  hipFree(igate_d);
  hipFree(hgate_d);
  hipFree(w_ih_enc_d);
  hipFree(w_hh_enc_d);
  hipFree(b_ih_enc_d);
  hipFree(b_hh_enc_d);
  hipFree(w_ih_dec_d);
  hipFree(w_hh_dec_d);
  hipFree(b_ih_dec_d);
  hipFree(b_hh_dec_d);
  hipFree(w_ho_d);

  hipHostFree(output);
  free(w_ih_enc);
  free(w_hh_enc);
  free(b_ih_enc);
  free(b_hh_enc);
  free(w_ih_dec);
  free(w_hh_dec);
  free(b_ih_dec);
  free(b_hh_dec);
  free(w_ho);
  free(emb_tbl_enc);
  free(emb_tbl_dec);

  return 0;
}

void batch_matmul(float *A, float *B, float *C, int bsz, int M, int N, int K) {
  if (bsz == 1 && M == 2048 && N == 1 && K == 512) {
    dim3 gridDim(1, 256, 1);
    dim3 blockDim(1, 2, 1);
    batch_matmul_1_2048_1_512_kernel<<<gridDim, blockDim>>>(A, B, C);
  } else if (bsz == 1 && M == 2048 && N == 1 && K == 256) {
    dim3 gridDim(1, 256, 1);
    dim3 blockDim(1, 8, 1);
    batch_matmul_1_2048_1_512_kernel<<<gridDim, blockDim>>>(A, B, C);
  } else if (bsz == 1 && M == 16384 && N == 1 && K == 512) {
    dim3 gridDim(1, 256, 1);
    dim3 blockDim(1, 16, 1);
    batch_matmul_1_16384_1_512_kernel<<<gridDim, blockDim>>>(A, B, C);
  } else if (bsz == 1 && M == 256 && N == 1 && K == 512) {
    dim3 gridDim(1, 4, 1);
    dim3 blockDim(1, 32, 1);
    batch_matmul_1_256_1_512_kernel<<<gridDim, blockDim>>>(A, B, C);
  } else if (bsz == 64 && M == 2048 && N == 1 && K == 512) {
    dim3 gridDim(1, 256, 1);
    dim3 blockDim(1, 2, 1);
    batch_matmul_64_2048_1_512_kernel<<<gridDim, blockDim>>>(A, B, C);
  } else if (bsz == 1 && M == 1 && N == 2048 && K == 512) {
    dim3 gridDim(64, 1, 1);
    dim3 blockDim(8, 1, 1);
    batch_matmul_1_1_2048_512_kernel<<<gridDim, blockDim>>>(A, B, C);
  } else if (bsz == 1 && M == 1 && N == 16384 && K == 512) {
    dim3 gridDim(512, 1, 1);
    dim3 blockDim(32, 1, 1);
    batch_matmul_1_1_16384_512_kernel<<<gridDim, blockDim>>>(A, B, C);
  } else if (bsz == 1 && M == 4 && N == 2048 && K == 512) {
    dim3 gridDim(256, 1, 1);
    dim3 blockDim(8, 4, 1);
    batch_matmul_1_4_2048_512_kernel<<<gridDim, blockDim>>>(A, B, C);
  } else if (bsz == 1 && M == 4 && N == 16384 && K == 512) {
    dim3 gridDim(256, 1, 1);
    dim3 blockDim(32, 1, 1);
    batch_matmul_1_4_16384_512_kernel<<<gridDim, blockDim>>>(A, B, C);
  } else if (bsz == 1 && M == 64 && N == 2048 && K == 512) {
    dim3 gridDim(64, 2, 1);
    dim3 blockDim(16, 2, 1);
    batch_matmul_1_64_2048_512_kernel<<<gridDim, blockDim>>>(A, B, C);
  } else if (bsz == 1 && M == 64 && N == 16384 && K == 512) {
    dim3 gridDim(256, 1, 1);
    dim3 blockDim(32, 2, 1);
    batch_matmul_1_64_16384_512_kernel<<<gridDim, blockDim>>>(A, B, C);
  } else {
    printf("batch_matmul: WRONG ARGS (bsz=%d, M=%d, N=%d, K=%d)", bsz, M, N, K);
    exit(-1);
  }
}
__device__ void batch_matmul_dev(float *A, float *B, float *C, int bsz, int M,
                                 int N, int K) {
  if (bsz == 1 && M == 2048 && N == 1 && K == 512) {
    dim3 gridDim(1, 256, 1);
    dim3 blockDim(1, 2, 1);
    batch_matmul_1_2048_1_512_kernel<<<gridDim, blockDim>>>(A, B, C);
  } else if (bsz == 1 && M == 2048 && N == 1 && K == 256) {
    dim3 gridDim(1, 256, 1);
    dim3 blockDim(1, 8, 1);
    batch_matmul_1_2048_1_512_kernel<<<gridDim, blockDim>>>(A, B, C);
  } else if (bsz == 1 && M == 16384 && N == 1 && K == 512) {
    dim3 gridDim(1, 256, 1);
    dim3 blockDim(1, 16, 1);
    batch_matmul_1_16384_1_512_kernel<<<gridDim, blockDim>>>(A, B, C);
  } else if (bsz == 1 && M == 256 && N == 1 && K == 512) {
    dim3 gridDim(1, 4, 1);
    dim3 blockDim(1, 32, 1);
    batch_matmul_1_256_1_512_kernel<<<gridDim, blockDim>>>(A, B, C);
  } else if (bsz == 64 && M == 2048 && N == 1 && K == 512) {
    dim3 gridDim(1, 256, 1);
    dim3 blockDim(1, 2, 1);
    batch_matmul_64_2048_1_512_kernel<<<gridDim, blockDim>>>(A, B, C);
  } else if (bsz == 1 && M == 1 && N == 2048 && K == 512) {
    dim3 gridDim(64, 1, 1);
    dim3 blockDim(8, 1, 1);
    batch_matmul_1_1_2048_512_kernel<<<gridDim, blockDim>>>(A, B, C);
  } else if (bsz == 1 && M == 1 && N == 16384 && K == 512) {
    dim3 gridDim(512, 1, 1);
    dim3 blockDim(32, 1, 1);
    batch_matmul_1_1_16384_512_kernel<<<gridDim, blockDim>>>(A, B, C);
  } else if (bsz == 1 && M == 4 && N == 2048 && K == 512) {
    dim3 gridDim(256, 1, 1);
    dim3 blockDim(8, 4, 1);
    batch_matmul_1_4_2048_512_kernel<<<gridDim, blockDim>>>(A, B, C);
  } else if (bsz == 1 && M == 4 && N == 16384 && K == 512) {
    dim3 gridDim(256, 1, 1);
    dim3 blockDim(32, 1, 1);
    batch_matmul_1_4_16384_512_kernel<<<gridDim, blockDim>>>(A, B, C);
  } else if (bsz == 1 && M == 64 && N == 2048 && K == 512) {
    dim3 gridDim(64, 2, 1);
    dim3 blockDim(16, 2, 1);
    batch_matmul_1_64_2048_512_kernel<<<gridDim, blockDim>>>(A, B, C);
  } else if (bsz == 1 && M == 64 && N == 16384 && K == 512) {
    dim3 gridDim(256, 1, 1);
    dim3 blockDim(32, 2, 1);
    batch_matmul_1_64_16384_512_kernel<<<gridDim, blockDim>>>(A, B, C);
  } else {
    printf("batch_matmul: WRONG ARGS (bsz=%d, M=%d, N=%d, K=%d)", bsz, M, N, K);
  }
}