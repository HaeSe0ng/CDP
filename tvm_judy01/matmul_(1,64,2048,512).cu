
#include <hip/hip_runtime.h>
extern "C" __global__ void default_function_kernel0(void* __restrict__ A, void* __restrict__ B, void* __restrict__ compute) {
  float compute_local[32];
  __shared__ float A_shared[1024];
  __shared__ float B_shared[256];
  float A_shared_local[32];
  float B_shared_local[1];
  for (int i_c_init = 0; i_c_init < 32; ++i_c_init) {
    compute_local[(i_c_init)] = 0.000000e+00f;
  }
  for (int k_outer = 0; k_outer < 32; ++k_outer) {
    __syncthreads();
    #pragma unroll
    for (int ax1_inner = 0; ax1_inner < 32; ++ax1_inner) {
      A_shared[((((((int)threadIdx.y) * 512) + (ax1_inner * 16)) + ((int)threadIdx.x)))] = ((float*)A)[(((((((int)threadIdx.y) * 16384) + (ax1_inner * 512)) + (k_outer * 16)) + ((int)threadIdx.x)))];
    }
    #pragma unroll
    for (int ax1_inner1 = 0; ax1_inner1 < 8; ++ax1_inner1) {
      B_shared[((((((int)threadIdx.y) * 128) + (ax1_inner1 * 16)) + ((int)threadIdx.x)))] = ((float*)B)[((((((((int)blockIdx.x) * 8192) + (((int)threadIdx.y) * 4096)) + (ax1_inner1 * 512)) + (k_outer * 16)) + ((int)threadIdx.x)))];
    }
    __syncthreads();
    for (int k_inner = 0; k_inner < 16; ++k_inner) {
      #pragma unroll
      for (int ax1 = 0; ax1 < 32; ++ax1) {
        A_shared_local[(ax1)] = A_shared[((((((int)threadIdx.y) * 512) + (ax1 * 16)) + k_inner))];
      }
      B_shared_local[(0)] = B_shared[(((((int)threadIdx.x) * 16) + k_inner))];
      #pragma unroll
      for (int i_c = 0; i_c < 32; ++i_c) {
        compute_local[(i_c)] = (compute_local[(i_c)] + (A_shared_local[(i_c)] * B_shared_local[(0)]));
      }
    }
  }
  #pragma unroll
  for (int i_inner_inner = 0; i_inner_inner < 32; ++i_inner_inner) {
    ((float*)compute)[(((((((int)threadIdx.y) * 65536) + (i_inner_inner * 2048)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)))] = compute_local[(i_inner_inner)];
  }
}

